#include <stdio.h>
#include <hip/hip_runtime.h>

float f(float);

__global__ void Dev_trap(
		const float a        /*in*/,
		const float b        /*in*/,
		const float h        /*in*/,
		const int   n        /*in*/,
		float*      trap_p   /*in/out*/){

	int my_i = blockDim.x * blockIdx.x + threadIdx.x; 

	/*f(x_0) and f(x_n) were computed on the host. So compute
	 f(x_1), f(x_2), ..., f(x_(n-1))*/

	if (0 < my_i && my_i < n){
		float my_x = a + my_i*h;
		float my_trap = my_x*my_x + 4.0;
		atomicAdd(trap_p, my_trap);
	}


} /*Dev_trap*/


/*Host code*/

void Trap_wrapper(
		const float  a             /* in */, 
		const float  b             /* in */,
		const int    n             /* in */,
		float*       trap_p        /* out */,
		const int    blk_ct        /* in */, 
		const int    th_per_blk    /* in */){

	//float f(float);

	/*trap_p storage allocated with cudaMallocManaged*/

	//cudaMallocManaged(&trap_p, sizeof(float));

	*trap_p = 0.5*(f(a)+f(b));
	float h = (b-a)/n;
	
	//printf("The result is %f\n", *trap_p);

    Dev_trap<<<blk_ct, th_per_blk>>>(a, b, h, n, trap_p);
	hipDeviceSynchronize();

	*trap_p = h*(*trap_p);

	printf("The result is %f\n", *trap_p);
	
}/* Trap_wrapper */


void Get_args(
        	const int argc         /*in*/,
	        char*     argv[]       /*in*/,
	        int*      n_p          /*out*/,
	        int*      blk_ct_p     /*out*/,
	        int*      th_per_blk_p /*out*/,
	        float*    a_p          /*out*/,
			float*    b_p          /*out*/){

	if (argc != 6){
		/*print an error message and exit*/

		printf("Error: argc must be six");
	}
    
	*n_p = strtol(argv[1], NULL, 10);
	*blk_ct_p = strtol(argv[2], NULL, 10);
	*th_per_blk_p = strtol(argv[3], NULL, 10);
	*a_p = strtol(argv[4], NULL, 10);
	*b_p = strtol(argv[5], NULL, 10);


	/*Is n > total thread count = blk_ct * th_per_blk?*/
	if (*n_p >(*blk_ct_p)*(*th_per_blk_p) ){

		/*print an error message and exit */

		printf("Error: number of partitions must be less than total threads used."); 
	}

} /*Get_args*/


float f(float x){
	return x*x + 4.0;
}


float Serial_trap(
	const float a /* in */,
	const float b /* in */,
	const int   n /* in*/){
		
	float x, h = (b-a)/n;
	float trap = 0.5*(f(a)+f(b));

	for (int i = 1; i <= n-1; i++){
		x = a + i*h;
		trap += f(x);
	}
	trap = trap*h;

	return trap;

}/*Serial_trap*/


int main(int argc, char* argv[]){

	int n, th_per_blk, blk_ct;

	float a, b;	

	float* trap_p;

	hipMallocManaged(&trap_p, sizeof(float));
	
	/*Ge the command line arguments*/
	Get_args(argc, argv, &n, &blk_ct, &th_per_blk, &a, &b);

	/*Call the wrapper function*/
	Trap_wrapper(a, b, n, trap_p, blk_ct, th_per_blk); 

	/*call Serial_trap */
	float serial = Serial_trap(a, b, n);

	float error = serial - (*trap_p);

	//printf("The error is %f\n", *trap_p);

	return 0; 

} /*main*/

